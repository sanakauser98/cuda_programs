#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matmul(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void initialize_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    int M = 1024, N = 1024, K = 1024; 

    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);

    float *h_A = (float*)malloc(sizeA);
    float *h_B = (float*)malloc(sizeB);
    float *h_C = (float*)malloc(sizeC);

    initialize_matrix(h_A, M, K);
    initialize_matrix(h_B, K, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matmul<<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "Computation complete!" << std::endl;

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}

